#include "hip/hip_runtime.h"
#include<iostream>
#include<opencv2/highgui/highgui.hpp>
#include<opencv2/imgproc/imgproc.hpp>
 
using namespace std;
using namespace cv;

#define NoOfBins 65536
#define maxThresold 10
#define minThersold 10

int histogram[NoOfBins];
unsigned char histogram_LUT[NoOfBins];
unsigned char *d_histogram_ptr;
unsigned int *d_hist_max, *d_hist_min;

__global__ void creatLUT(unsigned char *d_histogram, unsigned int *hist_min, unsigned int *hist_max)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    float min_max_diff = *hist_max - *hist_min;
    int d_NoOfBins = 65536; 
    if (idx < d_NoOfBins)
    {
        float new_pixel = (idx - *hist_min)/min_max_diff;
        if(idx >= *hist_max)
        {
            new_pixel = 1;
        }
        else if(idx <= *hist_min)
        {
            new_pixel = 0;
        }
        d_histogram[idx] = (unsigned char)(new_pixel*255);
    }
}

__global__ void applyAGC(unsigned short *src_img, unsigned char  *proc_image, unsigned char *d_histogram, int img_rows, int img_cols)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

     proc_image[y*img_cols + x] = d_histogram[src_img[y*img_cols + x]];
}
 
int main()
{
    // Mat image = imread("nature.jpg", 0);
    Mat image = imread("images/frameIndex_0.png", -1);

    if( image.empty() )
    {
        cout << "Image not Found" << endl;
        return EXIT_FAILURE;
    }
    Mat proc_image = Mat::zeros(Size(image.cols,image.rows),CV_8UC1);

    // Create two temporary images (for holding sobel gradients)
    unsigned char *d_process_img;
    unsigned short *d_original_image;
    hipMalloc(&d_original_image, image.cols * image.rows* sizeof(unsigned short));
    hipMalloc(&d_process_img, image.cols* image.rows* sizeof(unsigned char));
    hipMalloc(&d_histogram_ptr, NoOfBins);

    hipMemcpy(d_original_image, image.data, image.rows * image.cols* sizeof(unsigned short), hipMemcpyHostToDevice);

    // allcoate memory for no of pixels for each intensity value
    /*     The maximum number of pixels can be total number of pixels in image.

    Total number of pixels in image resolution 640x512 is = 327680 

    The number of bins in 16 bit image is => 2^16 = 65536
    */

 
    // initialize all intensity values to 0
    for(int i = 0; i < NoOfBins; i++)
    {
        histogram[i] = 0;
    }
    
    // cout << sizeof(unsigned short)<< endl;
    cout << "pixel value:" << image.at<u_int16_t>(100,100)<< endl;
    cout <<" Channels:" << image.channels()<< endl;

    // calculate the no of pixels for each intensity values
    for(int y = 0; y < image.rows; y++)
    {
        for(int x = 0; x < image.cols; x++)
        {
            histogram[(int)image.at<u_int16_t>(y,x)]++;
        }
    }
 
    // draw the histograms
    int hist_w = 512; int hist_h = 400;
    // int bin_w = cvRound((double) hist_w/NoOfBins);
    double bin_w = (double) hist_w/NoOfBins;

 
    Mat histImage(hist_h, hist_w, CV_8UC1, Scalar(255, 255, 255));
 
     // find the maximum intensity element from histogram
    int hist_max = histogram[0];
    for(int i = 1; i < NoOfBins; i++){
        if(hist_max < histogram[i]){
            hist_max = histogram[i];
        }
    }
    // find the maximum intensity element from histogram
    int max = histogram[NoOfBins-1];
    for(int i = NoOfBins-2; i > 1; i--){
        if(maxThresold < histogram[i]){
            max = i;
            break;
        }
    }

    // find the minimum intensity element from histogram
    int min = histogram[1];
    for(int i = 2; i < NoOfBins; i++){
        if(minThersold < histogram[i]){
            min = i;
            break;
        }
    }
    // // Fill the LUT with new pixel value
    // float new_pixel, min_max_diff;
    // min_max_diff = max-min;

    // for(int i = 0; i < NoOfBins; i++)
    // {
    //         new_pixel = (i - min)/min_max_diff;
    //         if(i >= max)
    //         {
    //             new_pixel = 1;
    //         }
    //         else if(i <= min)
    //         {
    //             new_pixel = 0;
    //         }
    //         histogram_LUT[i] = (unsigned char)(new_pixel*255);
    // }
    
    // hipMemcpy(d_histogram_ptr, histogram_LUT, NoOfBins, hipMemcpyHostToDevice);


    cout << "max:" << max << endl << "min:" << min <<endl;

                // convolution kernel launch parameters
    dim3 cblocks (image.cols / 16, image.rows/ 16);
    dim3 cthreads(16, 16);

    // pythagoran kernel launch paramters
    dim3 pblocks (image.cols * image.rows / 256);
    dim3 pthreads(256, 1);

    hipMemset(d_histogram_ptr, 0, NoOfBins);
    hipMemcpy(d_hist_max, &max, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_hist_min, &min, sizeof(unsigned int), hipMemcpyHostToDevice);


    creatLUT<<<NoOfBins/256,256>>>(d_histogram_ptr, d_hist_min, d_hist_max);
    hipDeviceSynchronize();

    applyAGC<<<pblocks,pthreads>>>(d_original_image, d_process_img, d_histogram_ptr, 480, 640);
    hipDeviceSynchronize();
    hipMemcpy(proc_image.data,d_process_img, image.rows * image.cols* sizeof(unsigned char), hipMemcpyDeviceToHost);


    // normalize the histogram between 0 and histImage.rows
 
    for(int i = 0; i < NoOfBins; i++){
        histogram[i] = ((double)histogram[i]/hist_max)*histImage.rows;
    }
 
 
    // draw the intensity line for histogram
    for(int i = 0; i < NoOfBins; i++)
    {
        line(histImage, Point(cvRound(bin_w*(i)), hist_h),
                              Point(cvRound(bin_w*(i)), hist_h - histogram[i]),
             Scalar(0,0,0), 1, 8, 0);
    }
 

    // display histogram
    namedWindow("Intensity Histogram");
    imshow("Intensity Histogram", histImage);
 
    namedWindow("Image");
    imshow("Image", proc_image);
    waitKey();
    return 0;
}