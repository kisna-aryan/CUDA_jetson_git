#include "hip/hip_runtime.h"
#include<iostream>
#include<opencv2/highgui/highgui.hpp>
#include<opencv2/imgproc/imgproc.hpp>
 
using namespace std;
using namespace cv;

#define NoOfBins 65536
#define maxThresold 10
#define minThersold 10

int histogram[NoOfBins];
int *d_hostogram;

__global__ void creatLUT(unsigned char *d_histogram, unsigned int hist_min, unsigned int hist_max)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    float min_max_diff = hist_max - hist_min;
    int d_NoOfBins = 65536; 
    if (idx < d_NoOfBins)
    {
        float new_pixel = (idx - hist_min)/min_max_diff;
        if(idx >= hist_max)
        {
            new_pixel = 1;
        }
        else if(idx <= hist_min)
        {
            new_pixel = 0;
        }
        d_histogram[idx] = (unsigned char)(new_pixel*255);
    }
}

__global__ void applyAGC(unsigned short *src_img, unsigned char  *proc_image, unsigned char *d_histogram, int img_rows, int img_cols)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

     proc_image[y*img_cols + x] = d_histogram[src_img[y*img_cols + x]];
}
 
int main()
{
    // Mat image = imread("nature.jpg", 0);
    Mat image = imread("images/frameIndex_0.png", -1);

    if( image.empty() )
    {
        cout << "Image not Found" << endl;
        return EXIT_FAILURE;
    }
    Mat proc_image = Mat::zeros(Size(image.cols,image.rows),CV_8UC1);

    // Create two temporary images (for holding sobel gradients)
    unsigned char *process_img;
    unsigned short *original_image;
    hipMalloc(&original_image, image.cols * image.rows);
    hipMalloc(&process_img, image.cols* image.rows);

    hipMemset(dJunk, 0, sz);

    // allcoate memory for no of pixels for each intensity value
    /*     The maximum number of pixels can be total number of pixels in image.

    Total number of pixels in image resolution 640x512 is = 327680 

    The number of bins in 16 bit image is => 2^16 = 65536
    */

 
    // initialize all intensity values to 0
    for(int i = 0; i < NoOfBins; i++)
    {
        histogram[i] = 0;
    }
    
    // cout << sizeof(unsigned short)<< endl;
    cout << "pixel value:" << image.at<u_int16_t>(100,100)<< endl;
    cout <<" Channels:" << image.channels()<< endl;

    // calculate the no of pixels for each intensity values
    for(int y = 0; y < image.rows; y++)
    {
        for(int x = 0; x < image.cols; x++)
        {
            histogram[(int)image.at<u_int16_t>(y,x)]++;
        }
    }
 
    // draw the histograms
    int hist_w = 512; int hist_h = 400;
    // int bin_w = cvRound((double) hist_w/NoOfBins);
    double bin_w = (double) hist_w/NoOfBins;

 
    Mat histImage(hist_h, hist_w, CV_8UC1, Scalar(255, 255, 255));
 
     // find the maximum intensity element from histogram
    int hist_max = histogram[0];
    for(int i = 1; i < NoOfBins; i++){
        if(hist_max < histogram[i]){
            hist_max = histogram[i];
        }
    }
    // find the maximum intensity element from histogram
    int max = histogram[NoOfBins-1];
    for(int i = NoOfBins-2; i > 1; i--){
        if(maxThresold < histogram[i]){
            max = i;
            break;
        }
    }

    // find the minimum intensity element from histogram
    int min = histogram[1];
    for(int i = 2; i < NoOfBins; i++){
        if(minThersold < histogram[i]){
            min = i;
            break;
        }
    }

  
    cout << "max:" << max << endl << "min:" << min <<endl;

                // convolution kernel launch parameters
    dim3 cblocks (image.cols / 16, image.rows/ 16);
    dim3 cthreads(16, 16);

    // pythagoran kernel launch paramters
    dim3 pblocks (image.cols * image.rows / 256);
    dim3 pthreads(256, 1);



    applyAGC<<<pblocks,pthreads>>>(deviceGradientX, deviceGradientY, edgesDataDevice);

    // normalize the histogram between 0 and histImage.rows
 
    for(int i = 0; i < NoOfBins; i++){
        histogram[i] = ((double)histogram[i]/hist_max)*histImage.rows;
    }
 
 
    // draw the intensity line for histogram
    for(int i = 0; i < NoOfBins; i++)
    {
        line(histImage, Point(cvRound(bin_w*(i)), hist_h),
                              Point(cvRound(bin_w*(i)), hist_h - histogram[i]),
             Scalar(0,0,0), 1, 8, 0);
    }
 

    // display histogram
    namedWindow("Intensity Histogram");
    imshow("Intensity Histogram", histImage);
 
    namedWindow("Image");
    imshow("Image", proc_image);
    waitKey();
    return 0;
}